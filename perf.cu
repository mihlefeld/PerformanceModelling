#include "hip/hip_runtime.h"
#include <limits>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <cstdio>
#include <iostream>

#include "hipblas.h"
#include "perf.h"
#include "common.h"

// Device list of all possible exponents i and j, 2 wide and num_building_blocks high, stored row major
__constant__ float building_blocks[] = {
    0, 0,
    0, 1,
    0, 2,
    1./4, 0,
    1./3, 0,
    1./4, 1,
    1./3, 1,
    1./4, 2,
    1./3, 2,
    1./2, 0,
    1./2, 1,
    1./2, 2,
    2./3, 0,
    3./4, 0,
    2./3, 1,
    3./4, 1,
    4./5, 0,
    2./3, 2,
    3./4, 2,
    1, 0,
    1, 1,
    1, 2,
    5./4, 0,
    5./4, 1,
    4./3, 0,
    4./3, 1,
    3./2, 0,
    3./2, 1,
    3./2, 2,
    5./3, 0,
    7./4, 0,
    2, 0,
    2, 1,
    2, 2,
    9./4, 0,
    7./3, 0,
    5./2, 0,
    5./2, 1,
    5./2, 2,
    8./3, 0,
    11./4, 0,
    3, 0,
    3, 1
};

/*
 * Device list of all possible combinations
 * combinations are D*D large and represent how the terms should be added and multiplied together
 * they are stored in row major format and for every row the spot in the row indicates which term it represents
 * when the element is 1 it is present as part of the multiplicative row, if the whole row is 0, then the result
 * will be 0. All D rows get added together, this allows us to represent all possible combinations.
 * Over every hard-coded combination the add/multiply meaning is written as a comment.
 */
__constant__ unsigned char combinations[256];

/*
 * The end_indices group the combinations by the number of 0 rows. This is needed because hipblasSgelsBatched
 * can only solve systems with a matrix A of full rank, when a 0 row is present, the resulting A matrix has as many
 * 0 columns as 0 rows in the combination. So we need to seperate these possibilities.
 * The end index is always the first index that is no longer part of the group.
 */
int combinations_2d_end_indices[] {
    1, 4
};

unsigned char combinations_2d[] {
    // 1c: a*b
    1, 1,
    0, 0,

    // 2c: a + b
    0, 1,
    1, 0,

    // 2c: a*b + a
    1, 1,
    1, 0,

    // 3c: a*b + b
    1, 1,
    0, 1
};

int combinations_3d_end_indices[] {
    1, 11, 23
};

unsigned char combinations_3d[] {
    // 1c: x*y*z
    1, 1, 1,
    0, 0, 0,
    0, 0, 0,

    // 2c: x*y*z + x
    1, 1, 1,
    1, 0, 0,
    0, 0, 0,

    // 2c: x*y*z + y
    1, 1, 1,
    0, 1, 0,
    0, 0, 0,

    // 2c: x*y*z + z
    1, 1, 1,
    0, 0, 1,
    0, 0, 0,

    // 2c: x*y*z + x*y
    1, 1, 1,
    1, 1, 0,
    0, 0, 0,

    // 2c: x*y*z + y*z
    1, 1, 1,
    0, 1, 1,
    0, 0, 0,

    // 2c: x*y*z + x*z
    1, 1, 1,
    1, 0, 1,
    0, 0, 0,

    // 2c: x*y + z
    1, 1, 0,
    0, 0, 1,
    0, 0, 0,

    // 2c: x*z + y
    1, 0, 1,
    0, 1, 0,
    0, 0, 0,

    // 2c: x*z + x
    0, 1, 1,
    1, 0, 0,
    0, 0, 0,

    // 2c: y*z + x
    0, 1, 1,
    1, 0, 0,
    0, 0, 0,

    // 3c: x+y+z
    1, 0, 0,
    0, 1, 0,
    0, 0, 1,

    // 3c: x*y*z + x*y + z
    1, 1, 1,
    1, 1, 0,
    0, 0, 1,

    // 3c: x*y*z + y*z + x
    1, 1, 1,
    0, 1, 1,
    1, 0, 0,

    // 3c: x*y*z + x*z + y
    1, 1, 1,
    1, 0, 1,
    0, 1, 0,

    // 3c: x*y*z + x + y
    1, 1, 1,
    1, 0, 0,
    0, 1, 0,

    // 3c: x*y*z + x + z
    1, 1, 1,
    1, 0, 0,
    0, 0, 1,

    // 3c: x*y*z + y + z
    1, 1, 1,
    0, 1, 0,
    0, 0, 1,

    // 3c: x*y + z + y
    1, 1, 0,
    0, 0, 1,
    0, 1, 0,

    // 3c: x*y + z + x
    1, 1, 0,
    0, 0, 1,
    1, 0, 0,

    // 3c: x*z + y + x
    1, 0, 1,
    0, 1, 0,
    1, 0, 0,

    // 3c: y*z + x + y
    0, 1, 1,
    1, 0, 0,
    0, 1, 0,

    // 3c: y*z + x + z
    0, 1, 1,
    1, 0, 0,
    0, 0, 1,
};

// -1 indicates, that there are no combinations with that many 0 rows
int combinations_4d_end_indices[] {
    1, -1, -1, 2
};

unsigned char combinations_4d[] {
    // 1c: a*b*c*d
    1, 1, 1, 1,
    0, 0, 0, 0,
    0, 0, 0, 0,
    0, 0, 0, 0,

    // 4c: a + b + c + d
    1, 0, 0, 0,
    0, 1, 0, 0,
    0, 0, 1, 0,
    0, 0, 0, 1
};

int combinations_5d_end_indices[] {
    1, -1, -1, -1, 2
};

unsigned char combinations_5d[] {
    // 1c: a*b*c*d*e
    1, 1, 1, 1, 1,
    0, 0, 0, 0, 0,
    0, 0, 0, 0, 0,
    0, 0, 0, 0, 0,
    0, 0, 0, 0, 0,

    // 5c: a + b + c + d + e
    1, 0, 0, 0, 0,
    0, 1, 0, 0, 0,
    0, 0, 1, 0, 0,
    0, 0, 0, 1, 0,
    0, 0, 0, 0, 1
};

__device__ float* get_matrix_element_ptr(GPUMatrix m, int x, int y) {
    return (float*)((char*)m.elements + y * m.pitch) + x;
}

__device__ float get_matrix_element(GPUMatrix m, int x, int y) {
    float* pElement = (float*)((char*)m.elements + y * m.pitch) + x;
    return *pElement;
}

template<int D>
__device__ float evaluate_single(unsigned char *combination, float coef, float *ctps, float *params) {
    float prod = coef;
    // if the combination is 0 0 0, zero should be returned, instead of prod
    bool nonzero = false;
    for (int i = 0; i < D; i++) {
        nonzero |= combination[i];
        if (combination[i])
            prod *= pow(params[i], ctps[i*2]) * pow(log2(params[i]), ctps[i*2 + 1]);
    }
    return nonzero ? prod : 0;
}

template<int D>
__device__ float evaluate_multi(unsigned char *combination, float *coefs, float *ctps, float *params) {
    float result = coefs[0];
    for (int i = 0; i < D; i++) {
        result += evaluate_single<D>(&combination[i*D], coefs[i + 1], ctps, params);;
    }
    return result;
}

template<int D>
__device__ void get_data_from_indx(int idx, float *ctps, unsigned char **combination, Counts counts) {
    int combination_index = idx / counts.hpc;
    int mod_idx = idx  % counts.hpc;
    *combination = &combinations[combination_index * D * D];

    int r = counts.hpc;
    for (int i = D - 1; i >= 0; i--) {
        r/= counts.building_blocks;
        int ctpi = mod_idx / r;
        mod_idx = mod_idx % r;
        for (int j = 0; j < 2; j++) {
            ctps[i * 2 + j] = building_blocks[ctpi * 2 + j];
        }
    }
}

template<int D>
__global__ void __launch_bounds__(256) prepare_gels_batched(GPUMatrix measurements, Counts counts, Matrices mats, int swap_indx) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if(idx >= counts.hypotheses)
        return;

    float *A = &mats.A[idx * (measurements.height * (D + 1))];
    float *C = &mats.C[idx * measurements.height];
    mats.aps[idx] = A;
    mats.cps[idx] = C;
    // TODO: this really shouldn't be here, I don't even know why it's faster tbh
    __shared__ float sctps[512][2*D];
    float *ctps = sctps[threadIdx.x];
    unsigned char *combination;

    get_data_from_indx<D>(idx, ctps, &combination, counts);

    for (int i = 0; i < measurements.height; i++) {
        // first element in every row should be 1, since there's always a constant component
        A[i] = 1;
    }

    for (int i = 0; i < measurements.height; i++) {
        // TODO: seperate coordinates and values
        int ii = i == swap_indx ? (measurements.height - 1) : (i == measurements.height - 1 ? swap_indx : i);
        C[i] = get_matrix_element(measurements, D, ii);
    }

    // TODO: alternative kernel for more than 500 measurements
    float column_cache[500];
    for (int j = 0; j < D; j++) {
        for (int i = 0; i < measurements.height; i++) {
            int ii = i == swap_indx ? (measurements.height - 1) : (i == measurements.height - 1 ? swap_indx : i);
            column_cache[i] = evaluate_single<D>(&combination[D*j], 1, ctps, get_matrix_element_ptr(measurements, 0, ii));
        }
        for (int i = 0; i < measurements.height; i++) {
            A[(j + 1) * measurements.height + i] = column_cache[i];
        }
    }
}

__device__ float smape(float pred, float actual) {
    float abssum = abs(pred) + abs(actual);
    return abssum != 0 ? 200 * (abs(pred - actual) / abssum) : 0;
}

__device__ float rss(float pred, float actual) {
    return pow(pred - actual, 2);
}

template<int D>
__global__ void compute_costs(GPUMatrix measurements, Counts counts,
                              Matrices mats, Costs costs,
                              int validation_index) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if(idx >= counts.hypotheses)
        return;
    float ctps[2*D];
    float *coefs = &mats.C[idx * measurements.height];
    unsigned char *combination;
    get_data_from_indx<D>(idx, ctps, &combination, counts);

    // assume the validation measurement is the last row in the matrix
    int i = validation_index;
    float *row_ptr = get_matrix_element_ptr(measurements, 0, i);
    float actual = row_ptr[D];
    float predicted = evaluate_multi<D>(combination, coefs, ctps, row_ptr);

    costs.rss[idx] += rss(predicted, actual);
    costs.smape[idx] += smape(predicted, actual) / (measurements.height - 1);
}

template <int D>
__global__ void save_hypothesis(GPUHypothesis g_hypo, int idx, Counts counts, GPUMatrix measurements, Matrices mats, Costs costs) {
    float *coefs = &mats.C[idx * measurements.height];
    unsigned char *combination;
    get_data_from_indx<D>(idx, g_hypo.exponents, &combination, counts);
    *g_hypo.smape = costs.smape[idx];
    *g_hypo.rss = costs.rss[idx];
    for (int i = 0; i < D*D; i++) {
        g_hypo.combination[i] = combination[i];
    }
    for (int i = 0; i < D + 1; i++) {
        g_hypo.coefficients[i] = coefs[i];
    }
}

template<int D>
void solve(CublasStuff cbstuff, Counts counts, Matrices mats, const int *end_indices, int solve_count) {
    int start_index = 0;
    for (int j = 0; j < D; j++) {
        int end_index = end_indices[j];
        if (end_index == -1) continue;
        int combination_count = end_index - start_index;
        CUBLAS_CALL(hipblasSgelsBatched(
                cbstuff.handle,
                HIPBLAS_OP_N,
                solve_count, // height of Aarray
                j + 2, // width of Aarray and height of Carray
                1, // width of Carray
                mats.aps + (counts.hpc * start_index), // Aarray pointer
                cbstuff.lda, // lda >= max(1,m)
                mats.cps + (counts.hpc * start_index), // Carray pointer
                cbstuff.lda, // ldc >= max(1,m)
                &cbstuff.info,
                nullptr,
                combination_count * counts.hpc
        )
        )
        start_index = end_index;
    }
}

template<int D>
void find_hypothesis_templated(
        Counts counts,
        unsigned char *combinations_array,
        int *end_indices,
        const CPUMatrix &measurements
    )
{
    int block_size = 128;
    int grid_size = div_up(counts.hypotheses, block_size);
    int info;
    CublasStuff cbstuff = create_cublas_stuff(counts);
    Matrices mats = create_matrices(counts);
    Costs costs = create_costs(counts);
    GPUMatrix d_measurements = matrix_alloc_gpu(measurements.width, measurements.height);
    matrix_upload(measurements, d_measurements);
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(combinations), combinations_array, counts.combinations * D * D, 0, hipMemcpyHostToDevice))


    for (int i = 0; i < measurements.height - 1; i++) {
        prepare_gels_batched<D><<<grid_size, block_size>>>(d_measurements, counts, mats, i);

        solve<D>(cbstuff, counts, mats, end_indices, measurements.height - 1);

        compute_costs<D><<<grid_size, block_size>>>(d_measurements, counts, mats, costs, i);
    }

    prepare_gels_batched<D><<<grid_size, block_size>>>(d_measurements, counts,mats, measurements.height - 1);

    solve<D>(cbstuff, counts, mats, end_indices, measurements.height);

    int min_cost_idx;
    CUBLAS_CALL(hipblasIsamin(cbstuff.handle, counts.hypotheses, costs.smape, 1, &min_cost_idx));
    min_cost_idx -= 1;
    GPUHypothesis g_hypo = create_gpu_hypothesis(D);
    CPUHypothesis c_hypo = create_cpu_hypothesis(D);
    save_hypothesis<D><<<1, 1>>>(g_hypo, min_cost_idx, counts, d_measurements, mats, costs);
    c_hypo.download(g_hypo);
    c_hypo.print();

    CUDA_CALL(hipDeviceSynchronize());

    destroy_costs(costs);
    destroy_matrices(mats);
    destroy_cublas_stuff(cbstuff);
    destroy_gpu_hypothesis(g_hypo);
    destroy_cpu_hypothseis(c_hypo);
    matrix_free_gpu(d_measurements);
}

void find_hypothesis(const CPUMatrix &measurements) {
    hipblasHandle_t handle;
    Counts counts;
    int num_buildingblocks = 39;
    int dimensions = measurements.width-1;
    switch(dimensions) {
        case 2:
            counts = Counts(2, num_buildingblocks, 4, measurements.height);
            find_hypothesis_templated<2>(
                    counts,
                    combinations_2d,
                    combinations_2d_end_indices,
                    measurements
            );
            break;
        case 3:
            counts = Counts(3, num_buildingblocks, 23, measurements.height);
            find_hypothesis_templated<3>(
                    counts,
                    combinations_3d,
                    combinations_3d_end_indices,
                    measurements
            );

            break;
        case 4:

            break;
        case 5:

            break;

        default:
            std::cerr << "Finding hypothesis with dimensions " << dimensions << " is not supported!" << std::endl;
            exit(EXIT_FAILURE);
    }

    // TODO return the hypothesis
}

CublasStuff create_cublas_stuff(Counts counts) {
    CublasStuff cbstuff{};
    cbstuff.lda = counts.measurements;
    CUBLAS_CALL(hipblasCreate(&cbstuff.handle));
    return cbstuff;
}

Matrices create_matrices(Counts counts) {
    Matrices mats{};
    CUDA_CALL(hipMalloc(&mats.A, counts.hypotheses * counts.measurements * (counts.dim+1) * sizeof(float)))
    CUDA_CALL(hipMalloc(&mats.C, counts.hypotheses * counts.measurements * sizeof(float)))
    CUDA_CALL(hipMalloc(&mats.aps, counts.hypotheses * sizeof(float*)))
    CUDA_CALL(hipMalloc(&mats.cps, counts.hypotheses * sizeof(float*)))
    return mats;
}

Costs create_costs(Counts counts) {
    Costs costs{};
    CUDA_CALL(hipMalloc(&costs.rss, counts.hypotheses * sizeof(float)))
    CUDA_CALL(hipMalloc(&costs.smape, counts.hypotheses * sizeof(float)))
    CUDA_CALL(hipMemset(costs.rss, 0, counts.hypotheses * sizeof(float)))
    CUDA_CALL(hipMemset(costs.smape, 0, counts.hypotheses * sizeof(float)))
    return costs;
}

void destroy_cublas_stuff(CublasStuff cbstuff) {
    CUBLAS_CALL(hipblasDestroy(cbstuff.handle))
}

void destroy_matrices(Matrices mats) {
    CUDA_CALL(hipFree(mats.A))
    CUDA_CALL(hipFree(mats.C))
    CUDA_CALL(hipFree(mats.aps))
    CUDA_CALL(hipFree(mats.cps))
}

void destroy_costs(Costs costs) {
    CUDA_CALL(hipFree(costs.rss))
    CUDA_CALL(hipFree(costs.smape))
}

GPUHypothesis create_gpu_hypothesis(int d) {
    GPUHypothesis hypo{};
    hypo.d = d;
    CUDA_CALL(hipMalloc(&hypo.combination, d * d))
    CUDA_CALL(hipMalloc(&hypo.coefficients, (d + 1) * sizeof(float)))
    CUDA_CALL(hipMalloc(&hypo.exponents, 2 * d * sizeof(float)))
    CUDA_CALL(hipMalloc(&hypo.smape, sizeof(float)))
    CUDA_CALL(hipMalloc(&hypo.rss, sizeof(float)))
    return hypo;
}

CPUHypothesis create_cpu_hypothesis(int d) {
    CPUHypothesis hypo{};
    hypo.d = d;
    hypo.combination = new unsigned char[d * d];
    hypo.coefficients = new float[d + 1];
    hypo.exponents = new float[2 * d];
    return hypo;
}

void destroy_gpu_hypothesis(GPUHypothesis g_hypo) {
    CUDA_CALL(hipFree(g_hypo.combination))
    CUDA_CALL(hipFree(g_hypo.coefficients))
    CUDA_CALL(hipFree(g_hypo.exponents))
    CUDA_CALL(hipFree(g_hypo.smape))
    CUDA_CALL(hipFree(g_hypo.rss))
}

void destroy_cpu_hypothseis(CPUHypothesis c_hypo) {
    delete [] c_hypo.combination;
    delete [] c_hypo.coefficients;
    delete [] c_hypo.exponents;
}

size_t calculate_memory_usage(Counts counts) {
    size_t sof = sizeof(float);
    size_t size_costs = counts.hypotheses * 2 * sof;
    size_t size_mat_ptrs = counts.hypotheses * 2 * sizeof(float*);
    size_t size_a_matrix = counts.hypotheses * counts.measurements * (counts.dim + 1) * sof;
    size_t size_c_vector = counts.hypotheses * counts.measurements * sof;
    return size_costs + size_mat_ptrs + size_a_matrix + size_c_vector;
}

Counts::Counts(int dim, int building_blocks, int combinations, int measurements):
    dim(dim), building_blocks(building_blocks), combinations(combinations), measurements(measurements) {
    hpc = pow(building_blocks, dim);
    hypotheses = combinations * hpc;
    hipDeviceProp_t device_props{};
    int device;
    hipGetDevice(&device);
    hipGetDeviceProperties(&device_props, device);
    size_t vram_target = device_props.totalGlobalMem * 0.8;
    size_t vram_cost = calculate_memory_usage(*this);
    batches = ceil(vram_cost / (float) vram_target);
    batch_size = ceil(hypotheses / (float) batches);
}

void CPUHypothesis::download(GPUHypothesis g_hypo) {
    CUDA_CALL(hipMemcpy(combination, g_hypo.combination, g_hypo.d * g_hypo.d, hipMemcpyDeviceToHost))
    CUDA_CALL(hipMemcpy(coefficients, g_hypo.coefficients, (g_hypo.d + 1)*sizeof(float), hipMemcpyDeviceToHost))
    CUDA_CALL(hipMemcpy(exponents, g_hypo.exponents, g_hypo.d*2*sizeof(float), hipMemcpyDeviceToHost))
    CUDA_CALL(hipMemcpy(&smape, g_hypo.smape, sizeof(float), hipMemcpyDeviceToHost))
    CUDA_CALL(hipMemcpy(&rss, g_hypo.rss, sizeof(float), hipMemcpyDeviceToHost))
}

void CPUHypothesis::print() {
    std::cout << "-----------------------------------------------------------------" << std::endl;
    std::cout << "Hypothesis (SMAPE = " << smape << ", RSS = " << rss << ")" << std::endl;
    std::cout << "Coefficients:";
    for (int i = 0; i < d + 1; i++)
        std::cout << " " << coefficients[i];
    std::cout << "\nExponents:";
    for (int i = 0; i < 2*d; i+=2) {
        std::cout << " " << "(" << exponents[i] << ", " << exponents[i+1] << ")";
    }
    std::cout << "\nCombination:" << std::endl;
    for (int i = 0; i < d * d; i++) {
        std::cout << (int) combination[i] << ", ";
        if (i%d == 1)
            std::cout << std::endl;
    }
    std::cout << "-----------------------------------------------------------------" << std::endl;
}
