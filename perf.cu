#include "hip/hip_runtime.h"
#define DEBUG
#define DEBUG_IDX 0 * 59319
#include <limits>
#include <hip/hip_runtime.h>
#include <cstdio>
#include <iostream>

#include "hipblas.h"
#include "perf.h"
#include "common.h"

__constant__ float building_blocks[] = {
        0, 1,
        0, 2,
        1./4, 0,
        1./3, 0,
        1./4, 1,
        1./3, 1,
        1./4, 2,
        1./3, 2,
        1./2, 0,
        1./2, 1,
        1./2, 2,
        2./3, 0,
        3./4, 0,
        2./3, 1,
        3./4, 1,
        4./5, 0,
        2./3, 2,
        3./4, 2,
        1, 0,
        1, 1,
        1, 2,
        5./4, 0,
        5./4, 1,
        4./3, 0,
        4./3, 1,
        3./2, 0,
        3./2, 1,
        3./2, 2,
        5./3, 0,
        7./4, 0,
        2, 0,
        2, 1,
        2, 2,
        9./4, 0,
        7./4, 0,
        2, 0,
        2, 1,
        2, 2,
        9./4, 0,
        7./3, 0,
        5./2, 0,
        5./2, 1,
        5./2, 2,
        8./3, 0,
        11./4, 0,
        3, 0,
        3, 1
};

__constant__ unsigned char combinations[256];

int combinations_2d_column_counts[] {
    0, 1
};

unsigned char combinations_2d[] {
        1, 1,
        0, 0,

        0, 1,
        1, 0,

        1, 1,
        1, 0,

        1, 1,
        0, 1
};

// combination counts: element at index i determines where the combinations
// with i + 2 columns start if -1, the combinations using that many columns are not present

int combinations_3d_start_indices[] {
    0, 1, 11
};

unsigned char combinations_3d[] {
    // 1c: x*y*z
    1, 1, 1,
    0, 0, 0,
    0, 0, 0,

    // 2c: x*y*z + x
    1, 1, 1,
    1, 0, 0,
    0, 0, 0,

    // 2c: x*y*z + y
    1, 1, 1,
    0, 1, 0,
    0, 0, 0,

    // 2c: x*y*z + z
    1, 1, 1,
    0, 0, 1,
    0, 0, 0,

    // 2c: x*y*z + x*y
    1, 1, 1,
    1, 1, 0,
    0, 0, 0,

    // 2c: x*y*z + y*z
    1, 1, 1,
    0, 1, 1,
    0, 0, 0,

    // 2c: x*y*z + x*z
    1, 1, 1,
    1, 0, 1,
    0, 0, 0,

    // 2c: x*y + z
    1, 1, 0,
    0, 0, 1,
    0, 0, 0,

    // 2c: x*z + y
    1, 0, 1,
    0, 1, 0,
    0, 0, 0,

    // 2c: x*z + x
    0, 1, 1,
    1, 0, 0,
    0, 0, 0,

    // 2c: y*z + x
    0, 1, 1,
    1, 0, 0,
    0, 0, 0,

    // 3c: x+y+z
    1, 0, 0,
    0, 1, 0,
    0, 0, 1,

    // 3c: x*y*z + x*y + z
    1, 1, 1,
    1, 1, 0,
    0, 0, 1,

    // 3c: x*y*z + y*z + x
    1, 1, 1,
    0, 1, 1,
    1, 0, 0,

    // 3c: x*y*z + x*z + y
    1, 1, 1,
    1, 0, 1,
    0, 1, 0,

    // 3c: x*y*z + x + y
    1, 1, 1,
    1, 0, 0,
    0, 1, 0,

    // 3c: x*y*z + x + z
    1, 1, 1,
    1, 0, 0,
    0, 0, 1,

    // 3c: x*y*z + y + z
    1, 1, 1,
    0, 1, 0,
    0, 0, 1,

    // 3c: x*y + z + y
    1, 1, 0,
    0, 0, 1,
    0, 1, 0,

    // 3c: x*y + z + x
    1, 1, 0,
    0, 0, 1,
    1, 0, 0,

    // 3c: x*z + y + x
    1, 0, 1,
    0, 1, 0,
    1, 0, 0,

    // 3c: y*z + x + y
    0, 1, 1,
    1, 0, 0,
    0, 1, 0,

    // 3c: y*z + x + z
    0, 1, 1,
    1, 0, 0,
    0, 0, 1,
};

int combinations_4d_start_indices[] {
    0, -1, -1, 1
};

unsigned char combinations_4d[] {
    1, 1, 1, 1,
    0, 0, 0, 0,
    0, 0, 0, 0,
    0, 0, 0, 0,

    1, 0, 0, 0,
    0, 1, 0, 0,
    0, 0, 1, 0,
    0, 0, 0, 1
};

int combinations_5d_start_indices[] {
    0, -1, -1, -1, 1
};

unsigned char combinations_5d[] {
    1, 1, 1, 1, 1,
    0, 0, 0, 0, 0,
    0, 0, 0, 0, 0,
    0, 0, 0, 0, 0,
    0, 0, 0, 0, 0,

    1, 0, 0, 0, 0,
    0, 1, 0, 0, 0,
    0, 0, 1, 0, 0,
    0, 0, 0, 1, 0,
    0, 0, 0, 0, 1
};

__device__ float* get_matrix_element_ptr(GPUMatrix m, int x, int y) {
    return (float*)((char*)m.elements + y * m.pitch) + x;
}

__device__ float get_matrix_element(GPUMatrix m, int x, int y) {
    float* pElement = (float*)((char*)m.elements + y * m.pitch) + x;
    return *pElement;
}

__device__ void set_matrix_element(GPUMatrix m, int x, int y, float v) {
    float* pElement = (float*)((char*)m.elements + y * m.pitch) + x;
    *pElement = v;
}

template<int D>
__device__ float evaluate_single(unsigned char *combination, float coef, float *ctps, float *params) {
    float prod = coef;
    // if the combination is 0 0 0, zero should be returned, instead of prod
    bool nonzero = 0;
    for (int i = 0; i < D; i++) {
        nonzero |= combination[i];
        if (combination[i])
            prod *= pow(params[i], ctps[i*2]) * pow(log2(params[i]), ctps[i*2 + 1]);
    }
    return nonzero ? prod : 0;
}

// coefs has to be initialized with all 1s
// coefs needs to be D + 1 in size
template<int D>
__device__ float evaluate_multi(unsigned char *combination, float *coefs, float *ctps, float *params) {
    float result = coefs[0];
    for (int i = 0; i < D; i++) {
        result += evaluate_single<D>(&combination[i*D], coefs[i + 1], ctps, params);;
    }
    return result;
}

template<int D>
__device__ void get_data_from_indx(int idx, float *ctps, unsigned char **combination,
                                   int num_combinations, int num_buildingblocks, int num_hypothesis) {
    int div_ci = num_hypothesis / num_combinations;
    int combination_index = idx / div_ci;
    int mod_idx = idx  % div_ci;
    *combination = &combinations[combination_index * D * D];

    int r = pow(num_buildingblocks, D-1);
    for (int i = D - 1; i >= 0; i--) {
        int ctpi = mod_idx / r;
        mod_idx = mod_idx % r;
        for (int j = 0; j < 2; j++) {
            ctps[i * 2 + j] = building_blocks[ctpi * 2 + j];
        }
        r/= num_buildingblocks;
    }
}

template<int D>
__global__ void prepare_gels_batched(GPUMatrix measurements, int num_combinations, int num_buildingblocks, int num_hypothesis,
                                     float *amatrices, float *cmatrices, float **amptrs, float **cmptrs, int swap_indx) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if(idx >= num_hypothesis)
        return;

    float *amatrix = &amatrices[idx * (measurements.height * (D + 1))];
    float *cmatrix = &cmatrices[idx * measurements.height];
    amptrs[idx] = amatrix;
    cmptrs[idx] = cmatrix;
    float ctps[2*D];
    unsigned char *combination;

    get_data_from_indx<D>(idx, ctps, &combination, num_combinations, num_buildingblocks, num_hypothesis);

    for (int i = 0; i < measurements.height; i++) {
        // first element in every row should be 1, since there's always a constant component
        amatrix[i] = 1;
    }

    for (int i = 0; i < measurements.height; i++) {
        // TODO: seperate coordinates and values
        int ii = i == swap_indx ? (measurements.height - 1) : (i == measurements.height - 1 ? swap_indx : i);
        cmatrix[i] = get_matrix_element(measurements, D, ii);
    }

    for (int j = 0; j < D; j++) {
        for (int i = 0; i < measurements.height; i++) {
            int ii = i == swap_indx ? (measurements.height - 1) : (i == measurements.height - 1 ? swap_indx : i);
            // this value needs to be written into a giant list of matrices
            float y = evaluate_single<D>(&combination[D*j], 1, ctps, get_matrix_element_ptr(measurements, 0, ii));
            // danger danger, amatrix must be column major format
            amatrix[(j + 1) * measurements.height + i] = y;
        }
    }
}

template<int D>
__global__ void compute_costs(GPUMatrix measurements, int num_combinations, int num_buildingblocks, int num_hypothesis,
                              float *cmatrices, float *rss_costs, float *smape_costs) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if(idx >= num_hypothesis)
        return;
    float ctps[2*D];
    float *coefs = &cmatrices[idx * measurements.height];
    unsigned char *combination;
    get_data_from_indx<D>(idx, ctps, &combination, num_combinations, num_buildingblocks, num_hypothesis);

    float rss_cost = 0;
    float smape_cost = 0;

    // assume the validation measurement is the last row in the matrix
    int i = measurements.height - 1;
    float *row_ptr = get_matrix_element_ptr(measurements, 0, i);
    float actual = row_ptr[D];
    float predicted = evaluate_multi<D>(combination, coefs, ctps, row_ptr);
    rss_cost = pow(predicted - actual, 2);
    float abssum = (abs(predicted) + abs(actual));
    if (abssum != 0)
        smape_cost = abs(predicted - actual) / (abs(predicted) + abs(actual));

    // TODO: don't forget to set all errors to zero before calling the cost functions
    rss_costs[idx] += rss_cost;
    smape_costs[idx] += 200 * smape_cost / (measurements.height - 1);
}

template<int D>
void find_hypothesis_templated(
        int num_buildingblocks,
        int num_combinations,
        unsigned char *combinations_array,
        int *start_indices,
        const CPUMatrix &measurements
    )
{
    hipblasHandle_t handle;
    int info;
    int num_hypothesis = pow(num_buildingblocks, D) * num_combinations;
    int hypothesis_per_combination = num_hypothesis / num_combinations;
    int *dev_info_array;
    float *amatrices, *cmatrices, **amptrs, **cmptrs, *rss_costs, *smape_costs;
    GPUMatrix device_measurements = matrix_alloc_gpu(measurements.width, measurements.height);

    // download pointers for testing
    CPUMatrix A = matrix_alloc_cpu(measurements.height, D + 1);
    CPUMatrix X = matrix_alloc_cpu(1, D + 1);
    CPUMatrix C = matrix_alloc_cpu(1, measurements.height);

    // allocate and upload data
    matrix_upload(measurements, device_measurements);
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(combinations), combinations_array, num_combinations * D * D, 0, hipMemcpyHostToDevice))
    CUDA_CALL(hipMalloc(&rss_costs, num_hypothesis * sizeof(float)));
    CUDA_CALL(hipMalloc(&smape_costs, num_hypothesis * sizeof(float)));
    CUDA_CALL(hipMalloc(&dev_info_array, num_hypothesis * sizeof(int)));
    CUDA_CALL(hipMalloc(&amatrices, num_hypothesis * measurements.height * (D+1) * sizeof(float)))
    CUDA_CALL(hipMalloc(&cmatrices, num_hypothesis * measurements.height * sizeof(float)))
    CUDA_CALL(hipMalloc(&amptrs, num_hypothesis * sizeof(float*)))
    CUDA_CALL(hipMalloc(&cmptrs, num_hypothesis * sizeof(float*)))
    CUDA_CALL(hipMemset(rss_costs, 0, num_hypothesis * sizeof(float)))
    CUDA_CALL(hipMemset(smape_costs, 0, num_hypothesis * sizeof(float)))

    CUBLAS_CALL(hipblasCreate(&handle));

    for (int i = 0; i < measurements.height - 1; i++) {
        prepare_gels_batched<3><<<div_up(num_hypothesis, 512), 512>>>(
                device_measurements,
                num_combinations,
                num_buildingblocks,
                num_hypothesis,
                amatrices,
                cmatrices,
                amptrs,
                cmptrs,
                i
        );

        int previous_start_index = 0;
        for (int i = 0; i < D; i++) {
            int start_index = start_indices[i];
            if (start_index == -1) continue;
            int combination_count = start_index - previous_start_index + 1;
            CUBLAS_CALL(hipblasSgelsBatched(
                    handle,
                    HIPBLAS_OP_N,
                    measurements.height - 1, // height of Aarray
                    i + 2, // width of Aarray and height of Carray
                    1, // width of Carray
                    amptrs + (hypothesis_per_combination * start_index), // Aarray pointer
                    measurements.height, // lda >= max(1,m)
                    cmptrs + (hypothesis_per_combination * start_index), // Carray pointer
                    measurements.height, // ldc >= max(1,m)
                    &info,
                    dev_info_array,
                    combination_count * hypothesis_per_combination
                )
            )
            previous_start_index = start_index;
        }

        compute_costs<D><<<div_up(num_hypothesis, 512), 512>>>(device_measurements, num_combinations, num_buildingblocks,
                                                               num_hypothesis, cmatrices, rss_costs, smape_costs);
    }

    matrix_free_cpu(A);
    matrix_free_cpu(X);
    matrix_free_cpu(C);
    CUDA_CALL(hipFree(amatrices))
    CUDA_CALL(hipFree(cmatrices))
    CUDA_CALL(hipFree(amptrs))
    CUDA_CALL(hipFree(cmptrs))
    CUDA_CALL(hipFree(rss_costs))
    CUDA_CALL(hipFree(smape_costs))
    CUDA_CALL(hipFree(dev_info_array))
    matrix_free_gpu(device_measurements);
}

void find_hypothesis(const CPUMatrix &measurements) {
    hipblasHandle_t handle;
    int num_combinations;
    int num_buildingblocks = 39;
    int dimensions = measurements.width-1;
    switch(dimensions) {
        case 2:

            break;
        case 3:
            num_combinations = 23;
            find_hypothesis_templated<3>(
                    num_buildingblocks,
                    num_combinations,
                    combinations_3d,
                    combinations_3d_start_indices,
                    measurements
            );

            break;
        case 4:

            break;
        case 5:

            break;

        default:
            std::cerr << "Finding hypothesis with dimensions " << dimensions << " is not supported!" << std::endl;
            exit(EXIT_FAILURE);
    }

    // TODO return the hypothesis
}
